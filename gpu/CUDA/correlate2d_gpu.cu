
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void correlate2d_gpu_kernel(float* result, float* image, float* kernel, int image_width, int image_height, int kernel_width, int kernel_height) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        
        if (i < image_width - kernel_width + 1 && j < image_height - kernel_height + 1) {
            float sum = 0.0f;
            for (int ki = 0; ki < kernel_width; ki++) {
                for (int kj = 0; kj < kernel_height; kj++) {
                    sum += kernel[ki * kernel_width + kj] * image[(i + ki) * image_width + (j + kj)];
                }
            }
            result[i * (image_height - kernel_height + 1) + j] = sum;
        }
    }
    
    void correlate2d_gpu(float* result, float* image, float* kernel, int image_width, int image_height, int kernel_width, int kernel_height) {
        float* d_result;
        float* d_image;
        float* d_kernel;

        hipMalloc((void**)&d_result, (image_width - kernel_width + 1) * (image_height - kernel_height + 1) * sizeof(float));
        hipMalloc((void**)&d_image, image_width * image_height * sizeof(float));
        hipMalloc((void**)&d_kernel, kernel_width * kernel_height * sizeof(float));

        hipMemcpy(d_image, image, image_width * image_height * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_kernel, kernel, kernel_width * kernel_height * sizeof(float), hipMemcpyHostToDevice);

        dim3 blockSize(16, 16);
        dim3 gridSize((image_width - kernel_width + 1 + blockSize.x - 1) / blockSize.x, (image_height - kernel_height + 1 + blockSize.y - 1) / blockSize.y);

        correlate2d_gpu_kernel<<<gridSize, blockSize>>>(d_result, d_image, d_kernel, image_width, image_height, kernel_width, kernel_height);

        hipMemcpy(result, d_result, (image_width - kernel_width + 1) * (image_height - kernel_height + 1) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_result);
        hipFree(d_image);
        hipFree(d_kernel);
    }
}